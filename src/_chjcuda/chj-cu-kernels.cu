#include "hip/hip_runtime.h"
// cudamatrix/bd-cu-kernels.cu

// In this file is the CUDA code of the CUDA kernels, plus the ANSI-C wrappers

#include <cfloat>
#include "chj-cu-kernels-ansi.h"
#include <cmath>

//static is not support
template<typename Real>
__device__
inline Real LogAdd(Real x, Real y) {
  Real diff;
  if (x < y) {
    diff = x - y;
    x = y;
  } else {
    diff = y - x;
  }
  // diff is negative.  x is now the larger one.

  if (diff >= -15.9424) {
    Real res;
    res = x + log1pf(expf(diff));
    return res;
  } else {
    return x;  // return the larger one.
  }
}

template<typename Real>
__global__
static void _chj_cuda_mat_add_mat(MatrixDim A_dim,Real * A,Real *B,Real *C,Real NOT_USE_VALUE){
	int i = blockIdx.x * blockDim.x + threadIdx.x; // row index
	int j = blockIdx.y * blockDim.y + threadIdx.y; // column index
	int index = i * A_dim.stride + j;
	if (i < A_dim.rows && j < A_dim.cols) {
		if(B[index] == NOT_USE_VALUE || C[index] == NOT_USE_VALUE ){
			A[index] = NOT_USE_VALUE;
		}else{
			A[index] = B[index] + C[index];
		}
	}
}//_chj_cuda_mat_add_mat

template<typename Real>
__global__
static void _chj_cuda_ctc_loss_fun(MatrixDim A_dim,Real * A,MatrixDim B_dim,Real *B,MatrixDim C_dim,Real *C,Real p,Real NOT_USE_VALUE){
	int i = blockIdx.x * blockDim.x + threadIdx.x; // row index
	int j = blockIdx.y * blockDim.y + threadIdx.y; // column index
	int index = i * A_dim.stride + j;
//B_dim not A_dim 这是因为我的程序要求
	if (i < B_dim.rows && j < A_dim.cols) {
		Real a=NOT_USE_VALUE;
		int n=C[j*C_dim.stride];
		for(int ii=1;ii<=n;ii++){
            int idb=i*B_dim.stride+ C[ j*C_dim.stride + ii]; //*** 
			if(B[idb]!=NOT_USE_VALUE){
				if(a==NOT_USE_VALUE)a=B[idb];
				else a=LogAdd(a,B[idb]);
			}
		}
		if(a!=NOT_USE_VALUE){
			A[index] -= expf(a-p); //7.29			
		}
	}
}//_chj_cuda_mat_add_mat

/***********************************************************************
 * ANSI-C wrappers of CUDA kernels
 */

void chj_cudaF_mat_add_mat(dim3 Gr,dim3 Bl,MatrixDim A_dim,float *A,float *B,float *C,float NOT_USE_VALUE){
	_chj_cuda_mat_add_mat<<<Gr,Bl>>>(A_dim,A,B,C,NOT_USE_VALUE);
}

void chj_cudaF_ctc_loss_fun(dim3 Gr,dim3 Bl,MatrixDim A_dim,float *A,MatrixDim B_dim,float *B,MatrixDim C_dim,float *C,float p,float NOT_USE_VALUE){
	_chj_cuda_ctc_loss_fun<<<Gr,Bl>>>(A_dim,A,B_dim,B,C_dim,C,p,NOT_USE_VALUE);
}



